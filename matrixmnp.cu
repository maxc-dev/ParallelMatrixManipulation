﻿#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


#define BLOCK_SIZE 32

// Matrices are stored in row-major order
typedef struct {
	int width;
	int height;
	float* elements;
} Matrix;

__global__ void MatrixMultKern(const Matrix A, const Matrix B, const Matrix C) {
	// Calculate the column index of C and B
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	// Calculate the row index of C and of A
	int row = blockIdx.y * blockDim.y + threadIdx.y;

	if ((row < A.height) && (col < B.width)) {
		float Cvalue = 0;
		// each thread computes one element of the block sub-matrix

		for (int k = 0; k < A.width; ++k) {
			Cvalue += A.elements[row * A.width + k] * B.elements[k * B.width + col];
		}
		C.elements[row * C.width + col] = Cvalue;
	}
}

// Matrix multiplication - Host Code
// Matrix dimensions are assumed to be multiples of BLOCK_SIZE
void MatrixMult(const Matrix h_A, const Matrix h_B, Matrix h_C) {
	// Load A and B into device memory
	Matrix d_A;
	d_A.width = h_A.width; d_A.height = h_A.height;
	size_t size = h_A.width * h_A.height * sizeof(float);
	hipMalloc(&d_A.elements, size);
	hipMemcpy(d_A.elements, h_A.elements, size, hipMemcpyHostToDevice);

	Matrix d_B;
	d_B.width = h_B.width; d_B.height = h_B.height;
	size = h_B.width * h_B.height * sizeof(float);
	hipMalloc(&d_B.elements, size);
	hipMemcpy(d_B.elements, h_B.elements, size, hipMemcpyHostToDevice);

	// Allocate C in Device memory
	Matrix d_C;
	d_C.width = h_C.width; d_C.height = h_C.height;
	size = h_C.width * h_C.height * sizeof(float);
	hipMalloc(&d_C.elements, size);

	// Invoke Kernel
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid(d_B.width / dimBlock.x, d_A.height / dimBlock.y);
	MatrixMultKern<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);

	// Read C from Device to Host
	hipMemcpy(h_C.elements, d_C.elements, size, hipMemcpyDeviceToHost);
	// Free Device Memory
	hipFree(d_A.elements);
	hipFree(d_B.elements);
	hipFree(d_C.elements);
}

// Requires 3 integer args
int main(int argc, char* argv[]) {
	Matrix A, B, C;
	// Read Dimensions of A and B
	A.height = atoi(argv[1]);
	A.width = atoi(argv[2]);
	B.height = A.width;
	B.width = atoi(argv[3]);
	A.elements = (float*)malloc(A.width * A.height * sizeof(float));
	B.elements = (float*)malloc(B.width * B.height * sizeof(float));
	C.height = A.height;
	C.width = B.width;
	C.elements = (float*)malloc(C.width * C.height * sizeof(float));

	for (int i = 0; i < A.height; i++)
		for (int j = 0; j < A.width; j++)
			A.elements[i * A.width + j] = (float)(rand() % 3) + 1;
	for (int i = 0; i < B.height; i++)
		for (int j = 0; j < B.width; j++)
			B.elements[i * B.width + j] = (float)(rand() % 2) + 1;
	MatrixMult(A, B, C);

	for (int i = 0; i < A.height; i++) {
		for (int j = 0; j < A.width; j++)
			printf("%f ", A.elements[i * A.width + j]);
		printf("\n");
	}
	printf("\n");

	for (int i = 0; i < B.height; i++) {
		for (int j = 0; j < B.width; j++)
			printf("%f ", B.elements[i * B.width + j]);
		printf("\n");
	}
	printf("\n");

	for (int i = 0; i < C.height; i++) {
		for (int j = 0; j < C.width; j++)
			printf("%f ", C.elements[i * C.width + j]);
		printf("\n");
	}
	printf("\n");
	return 0;
}